#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits>
#include <stdint.h>

#define NUM_BLOCKS 28
#define THREADS_PER_BLOCK 1024
#define MAX 0xFFFFFFFFFFFFFFF
__global__ void myKernel() {
    if (threadIdx.x == 0) {
    }
    __syncthreads();
}

int main() {
    // int i=0;
    while (true){
        myKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
        // printf("%d",i++);
        hipDeviceSynchronize();
    }
    return 0;
}
