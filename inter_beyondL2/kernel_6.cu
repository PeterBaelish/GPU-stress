
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void kernel(float* data) {
    extern __shared__ float cache[];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int i = bid * blockDim.x + tid;

    // Load data into shared memory
    for (int j = tid; j < 3145728; j += blockDim.x) {
        cache[j] = data[i + j];
    }

    __syncthreads();

    // Process data
    for (int j = tid; j < 3145728; j += blockDim.x) {
        cache[j] += 1.0f;
    }

    __syncthreads();

    // Write data back to global memory
    for (int j = tid; j < 3145728; j += blockDim.x) {
        data[i + j] = cache[j];
    }
}

int main() {
    // Allocate device memory
    float* d_data;
    hipMalloc((void**)&d_data, 18874368 * sizeof(float));

    // Initialize data on host
    float* h_data = (float*)malloc(18874368 * sizeof(float));
    for (int i = 0; i < 18874368; i++) {
        h_data[i] = i;
    }

    // Copy data to device
    hipMemcpy(d_data, h_data, 18874368 * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int block_size = 256;
    int num_blocks = 6;
    int shared_mem_size = 3145728 * sizeof(float);
    while(true){
        kernel<<<num_blocks, block_size, shared_mem_size>>>(d_data);
    }

    // Copy result back to host
    hipMemcpy(h_data, d_data, 18874368 * sizeof(float), hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 18874368; i++) {
        if (h_data[i] != i + 1.0f) {
            printf("Error: h_data[%d] = %f, expected %f\n", i, h_data[i], i + 1.0f);
            break;
        }
    }

    // Free memory
    hipFree(d_data);
    free(h_data);

    return 0;
}