
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void int32_t_mult(int32_t *a, int32_t *b, int32_t *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int smID;
    // 获取当前线程所在的SM ID
    // asm("mov.u32 %0, %smid;" : "=r"(smID));
    // printf("Thread %d is running on SM %d\n", threadIdx.x, smID);
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

int main(int argc, char** argv) {
    int n = 1024;
    size_t size = n * sizeof(int32_t);
    int32_t *h_a = (int32_t *)malloc(size);
    int32_t *h_b = (int32_t *)malloc(size);
    int32_t *h_c = (int32_t *)malloc(size);

    // 初始化输入数据
    for (int i = 0; i < n; i++) {
        h_a[i] = int32_t(i);
        h_b[i] = int32_t(2 * i);
    }

    int32_t *d_a = NULL;
    int32_t *d_b = NULL;
    int32_t *d_c = NULL;

    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }
    err=hipMalloc((void **)&d_b, size);
    if(err != hipSuccess){
        err=hipMalloc((void **)&d_b, size);
    }
    err=hipMalloc((void **)&d_c, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_c, size);
    }


    // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    }

    // 调用CUDA核函数
    int block_size = 32; //256
    int grid_size = 28; //(n + block_size - 1) / block_size;


    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }

    while(true){
        int32_t_mult<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
    }

    
    // 将计算结果从设备内存复制回主机内存
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


    // 释放内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}
            