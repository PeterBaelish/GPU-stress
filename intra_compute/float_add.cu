
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void float_add(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int smID;
    // 获取当前线程所在的SM ID
    // asm("mov.u32 %0, %smid;" : "=r"(smID));
    // printf("Thread %d is running on SM %d\n", threadIdx.x, smID);
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char** argv) {
    int n = 1024;
    size_t size = n * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // 初始化输入数据
    for (int i = 0; i < n; i++) {
        h_a[i] = float(i);
        h_b[i] = float(2 * i);
    }

    float *d_a = NULL;
    float *d_b = NULL;
    float *d_c = NULL;

    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }
    err=hipMalloc((void **)&d_b, size);
    if(err != hipSuccess){
        err=hipMalloc((void **)&d_b, size);
    }
    err=hipMalloc((void **)&d_c, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_c, size);
    }


    // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    }

    // 调用CUDA核函数
    int block_size = 32; //256
    int grid_size = 28; //(n + block_size - 1) / block_size;


    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }

    while(true){
        float_add<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
    }

    
    // 将计算结果从设备内存复制回主机内存
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


    // 释放内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}
            