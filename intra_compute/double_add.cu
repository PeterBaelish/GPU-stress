
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void double_add(double *a, double *b, double *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int smID;
    // 获取当前线程所在的SM ID
    // asm("mov.u32 %0, %smid;" : "=r"(smID));
    // printf("Thread %d is running on SM %d\n", threadIdx.x, smID);
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char** argv) {
    int n = 1024;
    size_t size = n * sizeof(double);
    double *h_a = (double *)malloc(size);
    double *h_b = (double *)malloc(size);
    double *h_c = (double *)malloc(size);

    // 初始化输入数据
    for (int i = 0; i < n; i++) {
        h_a[i] = double(i);
        h_b[i] = double(2 * i);
    }

    double *d_a = NULL;
    double *d_b = NULL;
    double *d_c = NULL;

    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }
    err=hipMalloc((void **)&d_b, size);
    if(err != hipSuccess){
        err=hipMalloc((void **)&d_b, size);
    }
    err=hipMalloc((void **)&d_c, size);
    if (err != hipSuccess){
        err=hipMalloc((void **)&d_c, size);
    }


    // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    }

    // 调用CUDA核函数
    int block_size = 32; //256
    int grid_size = 28; //(n + block_size - 1) / block_size;


    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }

    while(true){
        double_add<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
    }

    
    // 将计算结果从设备内存复制回主机内存
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


    // 释放内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}
            