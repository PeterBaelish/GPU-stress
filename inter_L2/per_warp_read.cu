
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void my_kernel(float *data) {
    
    int index = threadIdx.x + (blockIdx.x * blockDim.x * warpSize);
    
    
    float value = data[index];
    
}

int main(int argc, char** argv) {
    // 定义数组大小和步长
    const int array_size = 3 * 1024 * 1024; // 3MB
    const int stride = 128; // 行大小
    size_t size = array_size * sizeof(float);
    // 分配数组
    float *h_a=(float *)malloc(size);
    for (int i = 0; i < array_size; i++) {
        h_a[i] = 123;
    }
    float *d_a = NULL;
    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    while (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }
     // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    while (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    // 定义网格和块大小
    int block_size = 256;
    int grid_size = (array_size + (block_size * stride) - 1) / (block_size * stride);

    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }

    // 调用CUDA核函数
    while(true){
        my_kernel<<<grid_size, block_size>>>(d_a);
    }
    hipDeviceSynchronize();

    // 释放数组
    hipFree(d_a);
    free(h_a);
    return 0;
}