
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>  

// 全读，128KB array,128B/行，warp
using namespace std;
__global__ void all_write(int *a, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid<n){
        // printf("%d\n",a[tid]);
        a[tid]--;
        // printf("%d\n",a[tid]);
    }
}



int main(int argc, char** argv) {
    int n = 1024*32;
    size_t size = n * sizeof(int);
    int *h_a = (int *)malloc(size);
    // printf("分配h_a内存\n");
    // 初始化输入数据
    for (int i = 0; i < n; i++) {
        h_a[i] = int(i);
    }

    int *d_a = NULL;

    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    while (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }
    // printf("分配d_a内存\n");
    // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    while (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    // 调用CUDA核函数
    int block_size = 1024; //threads per block
    int grid_size = 28; //(n + block_size - 1) / block_size;

    // printf("解析命令行参数\n");
    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }
    // printf("开始all_write\n");

    while(true){
        all_write<<<grid_size, block_size>>>(d_a, n);
    }


    // 将计算结果从设备内存复制回主机内存
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);


    // 释放内存
    hipFree(d_a);
    free(h_a);
    return 0;
}
            