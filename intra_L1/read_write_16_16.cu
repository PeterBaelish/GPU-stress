
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// 全读，128KB array,128B/行，warp
using namespace std;
__global__ void read_write_16_16(int *a, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int val;
    if (tid<n){
        if(tid%2==0){
            val=a[tid];
            val++;
        }else{
            a[tid]++;
        }
        
    }
}

int main(int argc, char** argv) {
    int n = 1024*32;
    size_t size = n * sizeof(int);
    int *h_a = (int *)malloc(size);

    // 初始化输入数据
    for (int i = 0; i < n; i++) {
        h_a[i] = int(i);
    }

    int *d_a = NULL;

    hipError_t err = hipSuccess;
    // 分配内存
    err=hipMalloc((void **)&d_a, size);
    while (err != hipSuccess){
        err=hipMalloc((void **)&d_a, size);
    }

    // Copy  主机的数组内容 to the gpu arrays
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    while (err != hipSuccess){
        err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    }

    // 调用CUDA核函数
    int block_size = 1024; //threads per block
    int grid_size = 28; //(n + block_size - 1) / block_size;


    // 解析命令行参数
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--block-size") == 0) {
            i++;
            block_size = atoi(argv[i]);
        } else if (strcmp(argv[i], "--grid-size") == 0) {
            i++;
            grid_size = atoi(argv[i]);
        }
        
    }

    while(true){
        read_write_16_16<<<grid_size, block_size>>>(d_a, n);
    }

    
    // 将计算结果从设备内存复制回主机内存
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);


    // 释放内存
    hipFree(d_a);
    free(h_a);
    return 0;
}
            